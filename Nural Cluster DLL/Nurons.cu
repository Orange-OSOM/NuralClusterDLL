#include "hip/hip_runtime.h"
#ifndef NuronsC
#define NuronsC

//#include"pch.h"
#include<utility>

#include "Nurons.h"


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



#include <hiprand.h>
#include <time.h> 
#include <chrono>

//load

#include <fstream>
#include <sstream> 
#include "parse.h"


namespace STE {
    // predeff

    __global__ void updateKernal(STE::Nuron* nurons, connection* conections, sts stats, int* rng);

    __global__ void updateActivation(STE::Nuron* nurons, connection* conections, sts stats, int* rng);

    __global__ void updateConections(STE::Nuron* nurons, connection* conections, sts stats, int* rng);

    __global__ void SaveKernal(char* nuronDecription, unsigned int infoPerNuron, STE::sts stats, STE::Nuron* nurons, connection* conections);

    __global__ void LoadKernal(char* dev_filText, unsigned int nurBlock, STE::sts stats, STE::Nuron* nurons, connection* connections);

    __global__ void setSensKernal(Nuron* nurons, unsigned quant, unsigned* ids, bool sensor);

    __global__ void updateSensKernal(Nuron* nurons, unsigned quant, unsigned* dev_ids, bool* dev_data);

    __global__ void updateOutKernal(Nuron* nurons, unsigned quant, unsigned* ids, bool* data);

    __global__ void rad(int* data, unsigned quantity, unsigned range, int offset);

    //math for gpu
    int u(int x) {
        double offset = .5;//0<#<1
        return (int)((tanh(900 * (x + offset)) / 2) + .5 + offset);
    }

    __device__ int uDev(int x) {
        double offset = .5;//0<#<1
        return (int)((tanh(900 * (x + offset)) / 2) + .5 + offset);
    }
    //deffs

        //exporting func (public:)
    __declspec(dllexport) Nurons::Nurons(unsigned int maxCons, unsigned int nuronRange, dim3 vol, unsigned int shape) {
        
        stats.volume = vol;
        //set total number of nurons
        stats.totalNum = vol.x * vol.y * vol.z;
        //allocate space for all nuron pointers
        allNurons = (Nuron*)malloc(stats.totalNum * sizeof(Nuron));
        //allocate sace for all nuron conections
        allConections = (connection*)malloc(stats.totalNum * maxCons * sizeof(connection));
        //set the maximum range for each nuron
        stats.maxRange = nuronRange;
        stats.maxConnections = maxCons;
        
        //set all conections to the default no nuron
        connection Default;
        for (unsigned i = 0; i < stats.totalNum * maxCons; i++)
        {
            allConections[i] = Default;
        }

        // initalize a nuron at every location allocated in the nuron allocation setp************************this needs to be rewritten
        int x = 0;
        int y = 0;
        int z = 0;
        for (unsigned int i = 0; i < stats.totalNum; i++)
        {
            //i can gpu accelorate this
            /*
            * here nurons are allocated and initalized
            *
            * nurons are allocated with id = to the position in the allocatedmemory.
            * they also recievve the memory locaion of their conections(witch are heald in one array for eas  copping to the gpu ram later			*/

            // makes a cube

            x = i % stats.volume.x;//we modulo of i andthe volume retuns a reppeting pattern from 0 to 1 under the volume thus the whole of the volume when puit in sequence
            y = (i / stats.volume.x) % stats.volume.y; //deviding i by the volume in the x direction gives a repeeting pattern that when evver i is a multiple of x increses indicating a new y value
            z = (i / (stats.volume.x * stats.volume.y)) % stats.volume.z; //same consept for y but devide by the area of one z heaight (x*y) 
            //std::cout << "i:"<<i<<" x:" << x << " y:" << y << " z:"<<z << std::endl;
            allNurons[i] = Nuron(maxCons, dim3(x, y, z), i, allConections + (i * maxCons), stats.maxRange, stats.maxNuronTime, stats.maxSensitivity);
            //printf("i:%d\n  x:%d\n  y:%d\n  z:%d\n\n",i,x,y,z);
        }
        
    }
     
    /*::Nurons(std::string file) {
        load(this, file);
    }*/

    __declspec(dllexport)Nurons::~Nurons() {
        //deallocate thearray of pointers to the nurons and the nuron conections 
        free(allConections);
        free(allNurons);
    }

    bool __declspec(dllexport) Nurons::updateCluster() {
        return errors.push(update());
    }

    bool __declspec(dllexport) Nurons::loadCluster(std::string in) {
        return errors.push(load(in));
    };

    bool __declspec(dllexport) Nurons::saveCluster(std::string out) {
        return errors.push(save(out));
    };

    bool __declspec(dllexport) Nurons::setNuronSensor(unsigned quant, unsigned* ids, bool sensor) {
        return errors.push(setNuronSens(quant, ids, sensor));
    };

    bool __declspec(dllexport) Nurons::updateSensorData(unsigned quant, unsigned* ids, bool* data) {
        return errors.push(updateSens(quant, ids, data));
    }

    bool __declspec(dllexport) Nurons::updateOutData(unsigned quant, unsigned* ids, bool* data) {
        return errors.push(updateOut(quant, ids, data));
    }

    void __declspec(dllexport) Nurons::IoData(unsigned quant, bool** dataPtr, unsigned* ids) {
        for (unsigned i = 0; i < quant; i++)
        {
            dataPtr[i] = &allNurons[ids[i]].activation;
        }
    }
    
    void Nurons::setNuronSens(unsigned id) {
        if (id < stats.totalNum) {
            allNurons[id].sensor = true;
        }
    }

    bool Nurons::getNuronSens(unsigned id) {
        if (id < stats.totalNum) {
            return allNurons[id].sensor;
        }
    }
   //error check
    bool __declspec(dllexport) Nurons::checkErrors() {
        return errors.isErr();
    }

    void __declspec(dllexport) Nurons::readErrors() {
        errors.reedAll();
    }
    
    
    //private:

    //depreceated untill purpas is deffined
    /* 
    hipError_t Nurons::run() {
        //using keybord inpuit to force an exit, c is going to be set to the pressed key value;
        char c;
        std::cout << "press esc to exit! " << std::endl;
        //loop indefinatly
        hipError_t cudaStatus = hipSuccess;
        while (cudaStatus == hipSuccess && true)
        {
            // get key board inpuit 
            c = getch();
            //compair inpuit to excape key charvalue witch is 27, then break the loop
            if (c == 27) break;

            //this launches the update routien and grabbs the error code;
            cudaStatus = this->update();

            // check if a cuda error occured, if so exit loop;
            if (cudaStatus != hipSuccess) break;
        }

        // check if error occord 
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!");
            //***void errorHandler();
        }

        // reset device (deallocates vram??)
        //https://stackoverflow.com/questions/36012289/what-is-the-role-of-cudadevicereset-in-cuda
        //"Note that this function will reset the device immediately.", " is used to destroy a CUDA context, which means that all device allocations are removed."
        cudaStatus = hipDeviceReset();
        // check if reset was sucess fule;
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            //void errorHandler();
        }

        //**print exit status
        std::cout << "exited: " << std::endl;
        return cudaStatus;
    }
    */

     hipError_t Nurons::update() {//*
         //std::cout << "bob" << std::endl;
        //need to be re written to seporate read conetons and set actication
         
        //ptr to all nurond on gpu
        Nuron* dev_allNur = {};
        //ptr to all conectiond on the gpu
        connection* dev_allconects = {};
        //ptr to all nurons rng
        int* host_RNG;
        host_RNG = (int*)malloc((stats.totalNum * 4) * sizeof(int));
        int* dev_RNG = {};
        //cuda error vareable
        // Choose which GPU to run on, change this on a multi-GPU system.
        hipError_t cudaStatus = hipSetDevice(0);
        if (errors.isErr()) goto ErrorUpdate;
        
        // create a grid of threds with the quantity of all the nurons
        unsigned int theadsPerBlock = 1024;
        dim3 grid((stats.totalNum / theadsPerBlock) + 1, 1, 1);
        //dim3 grid(stats.totalNum, 1, 1);


        // delllet this---------------------------------------------------------------
       // dim3 RNG3(rand() % 100 + 900000, rand() % 100 + 900000, rand() % 100 + 900000);

   
        //cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice (update) failed!  Do you have a CUDA-capable GPU installed?");
            goto ErrorUpdate;
        }

        // allocate space on the gpu for all the nurons and set the dev_allNur ptr to that location
        cudaStatus = hipMalloc((void**)&dev_allNur, stats.totalNum * sizeof(STE::Nuron));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc (update) (nurons) failed!");
            goto ErrorUpdate;
        }

        // Copy nurons to device memory
        //                                                                  printf("!!!!!!!!!!!!!!!max connection cpu:%d\n", stats.maxConnections);
        //printf("!!!!!!!!!!!!!!!current connect:%d\n", allNurons[1].currnetConections);
        cudaStatus = hipMemcpy(dev_allNur, allNurons, stats.totalNum * sizeof(STE::Nuron), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (update) (nurons) failed!");
            goto ErrorUpdate;
        }

        //allocate meemory on device for all the connections
        cudaStatus = hipMalloc((void**)&dev_allconects, stats.totalNum * stats.maxConnections * sizeof(connection));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc (update) (conections) failed!");
            goto ErrorUpdate;
        }
       

        //coppy all cenections to device memory
        cudaStatus = hipMemcpy(dev_allconects, allConections, stats.totalNum * stats.maxConnections * sizeof(connection), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (update) (conections) failed!");
            goto ErrorUpdate;
        }
        
        //aalocate memory on gpu for rng and xyz  for possible new conections in this step
        cudaStatus = hipMalloc((void**)&dev_RNG, stats.totalNum * 4 * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc (update) (rng) failed!");
            goto ErrorUpdate;
        }        

        //genorate all random numbers needed for each nuron
            //this section can be more efficent currntly calling a functioin that makes allocates device memory puits the rng naumbers there and then returs it to host meory passes it back and then nit gets puit back in to device memory. in the fututre create a function that can be passed a device pointer from htere handle the data on the gpu
        //prints remove later
        //debug std::cout << "\nrand:x,y,z\n";

        cudaStatus = Rand(&host_RNG, stats.totalNum*3, (stats.maxRange*2)+1, -stats.maxRange);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (update) (rng) failed!");
            goto ErrorUpdate;
        }


        //debug: std::cout << "\nrand:prob\n";

        int* host_RNG_temp = &host_RNG[stats.totalNum * 3];

        cudaStatus = Rand(&host_RNG_temp, stats.totalNum, stats.maxConnections, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (update) (rng) failed!");
            goto ErrorUpdate;
        }

        cudaStatus = hipMemcpy(dev_RNG, host_RNG, stats.totalNum * 4 * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (update) (rng) failed!");
            goto ErrorUpdate;
        }

        /*for (unsigned i = 0; i < stats.totalNum * 4; i++)
        {
            printf("i: % d = %d\n", i, host_RNG[i]);
        }/**/

        // Launch a kernel on the GPU with one thread for each element.
        //std::cout << "testing" << std::endl;
        updateActivation <<< grid, theadsPerBlock >>> (dev_allNur, dev_allconects, stats, dev_RNG);//just added this 
        // Check for any errors launching the kernel

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel (update) launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto ErrorUpdate;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize (update) returned error code %d after launching addKernel!\n", cudaStatus);
            goto ErrorUpdate;
        }
        updateConections <<< grid, theadsPerBlock >>> (dev_allNur, dev_allconects, stats, dev_RNG);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel (update) launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto ErrorUpdate;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize (update) returned error code %d after launching addKernel!\n", cudaStatus);
            goto ErrorUpdate;
        }

        // Copy nurons from gpu memory to cpu memory.
        cudaStatus = hipMemcpy(allNurons, dev_allNur, stats.totalNum * sizeof(STE::Nuron), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (update) (Nurons) failed!");
            goto ErrorUpdate;
        }

        // Copy connections from gpu memory to cpu memory.
        cudaStatus = hipMemcpy(allConections, dev_allconects, stats.totalNum*stats.maxConnections * sizeof(connection), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy (update) (conections) failed!");
            goto ErrorUpdate;
        }
        //printf("no err");
        
    ErrorUpdate:
        hipFree(dev_allNur);
        hipFree(dev_allconects);
        hipFree(dev_RNG);
        free(host_RNG);

        if (cudaStatus != hipSuccess) {
            if(!errors.isErr())errors.push(cudaStatus);
        }
       
       

        return cudaStatus;
    }


    //the issue is in this section
    __global__ void updateActivation(STE::Nuron * nurons, connection * conections, sts stats, int* rng) {
        //initilization---------------------------------------------------
        unsigned int I = (blockIdx.x * blockDim.x) + threadIdx.x;
        //end start--------------------------------------------------
        //printf("upActiv:%d", I);
        int timeAdd = 2;
        int timeSub = 1;
        int sensAdd = 2;
        int sensSub = 1;

        //printf("rngX:%d\nrngY:%d\nrngZ:%d\nrngT:%d\n\n", rngX, rngY, rngZ, rngT);
        if (stats.totalNum <= I) return;//exit if excess
        //if (I != 0)return;
        nurons[I].connections = conections + (I * stats.maxConnections);//set id on gpu
        //if (I == 0)printf("1Device current connect:%d\n", nurons[1].currnetConections);

        //summ all conceted nuron conections----------------------------------------------------------------------------------
        int fialValue = 0;
        //edit this
        for (unsigned i = 0; i < stats.maxConnections; i++) // accelorate this
        {
            
            
            if (nurons[I].connections[i].connected)
            {
                //printf("nurons[%d].connections[%d].connected = %d\n", I, i, nurons[I].connections[i].connected);
                if (nurons[nurons[I].connections[i].id].activation)
                {
                    //printf("%i:_con:%i_NC_times:%i\n",I,i, nurons[I].connections[i].time);
                    fialValue = fialValue + (nurons[I].connections[i].time);
                    nurons[I].connections[i].time = nurons[I].connections[i].time + timeAdd;
                    if (nurons[I].connections[i].time > nurons[I].maxNuronTime)
                    {
                        nurons[I].connections[i].time = nurons[I].maxNuronTime;
                    }
                }
                else
                {
                    //decreemt time  left and check if time is zeroif so remove from list
                    nurons[I].connections[i].time = nurons[I].connections[i].time - timeSub;
                    //printf("%i:_con:%i_NC_times:%i\n",I,i, nurons[I].connections[i].time);
                    if (nurons[I].connections[i].time < 1) {
                        //printf("remove!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!");
                        nurons[I].connections[i].connected = false;
                        nurons[I].connections[i].time = 0;
                        nurons[I].connections[i].id = 0;
                        nurons[I].currnetConections--;
                    }
                }
            }
        }

        // activate or not-------------------------------------------------------------------------------------------
        if (!nurons[I].sensor)
        {
            if (fialValue > nurons[I].sensitivity) //if summation is more than its senitivity set true and increment sensitivity
            {
                nurons[I].activation = true;
                nurons[I].sensitivity += sensAdd;
            }
            else {
                nurons[I].activation = false;
                nurons[I].sensitivity -= sensSub;
            }

            if (nurons[I].sensitivity > nurons[I].maxSensitivity)
            {
                nurons[I].sensitivity = nurons[I].maxSensitivity;
            }

        }
        else {
           //do nothing, its activation should remain what was set during host process
            //should simplifie this after i finalize design
        }
    }

    __global__ void updateConections(STE::Nuron * nurons, connection * conections, sts stats, int* rng) {
        //initilization---------------------------------------------------
        //I will be referd to as the thred index but this is a simplification of of its actual meaning. 
        //nvidea gpus can compute a maximum number of theds at one time(this is diffrent per gpu (is allwas a multipule of 2)). 
        //and infact can not compute less or more.
        //nvidea handles computeing more threds by computing them in blocks. nvidea allows for theds, and blocks to be indexed 3 dimentonaly.
        // i chose to keep evrethingin one dimention for simplisity. 
        // in order to compute less threds than are in a block you siply exclude all threds beon then desierd index.
        unsigned int I = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (I >= stats.totalNum) {
          
            return;
        }
        else {
            nurons[I].connections = conections+I * stats.maxConnections;           
            
            //if location is out of bounds "reflect" it at the boundry
            int* rngX;
            int* rngY;
            int* rngZ;
            int* rngT;

            //rng contains all random numbers for each nuron. this includes rngx,rngy,rngz, and rng probability of new conection.
            //thees valus are stored in the single array in the order (example 2 nurons): x0,y0,z0,x1,y1,z1,p0,p1.
            //this pattern continues with more nurons
            //here rngx is set to the location in the array rng offset by (I is the thred index) I*3. for I = 0: first location of rng + 0
            //for I = 1: first location of rng + 3
            rngX = rng + (3 * I);
            rngY = rngX + 1; //here rngy is set to the location after rngx 
            rngZ = rngY + 1; //here rngz is set to the location after rngy

            // here rngT (the probability of the nuron forming a connection) is set to the first location in rng that has a location that contains a probability.
            // due to the way the random numbers are stored the first location is located at the first location of rng offset by 3* the total number of nurons.
            // this inital probabiliyy location must then be ofset by the index of the thred(I)
            rngT = rng + (3 * stats.totalNum) + I;

            unsigned xd = (uDev(*rngX) * *rngX) + (uDev(*rngX - stats.volume.x) * ((stats.volume.x - (*rngX % stats.volume.x) - 1) - *rngX)) + (uDev(-1 - *rngX) * (*rngX % stats.volume.x));
            unsigned yd = (uDev(*rngY) * *rngY) + (uDev(*rngY - stats.volume.y) * ((stats.volume.y - (*rngY % stats.volume.y) - 1) - *rngY)) + (uDev(-1 - *rngY) * (*rngY % stats.volume.y));
            unsigned zd = (uDev(*rngZ) * *rngZ) + (uDev(*rngZ - stats.volume.z) * ((stats.volume.z - (*rngZ % stats.volume.z) - 1) - *rngZ)) + (uDev(-1 - *rngZ) * (*rngZ % stats.volume.z));
            unsigned td = (uDev(*rngT) * *rngT) + (uDev(*rngT - stats.maxConnections) * ((stats.maxConnections - (*rngT % stats.maxConnections) - 1) - *rngT)) + (uDev(-1 - *rngT) * (*rngT % stats.maxConnections));

            
            //cpmpare a random number to the amount of un used connections----------------------------------------------------------------------------------------------------------

            //decide if nuron wil make connection.
            //if (*rngT > (nurons[I].currnetConections / stats.maxConnections) )
            //printf("%i < %u||%u||%u||%d\n", td, stats.maxConnections - nurons[I].currnetConections, stats.maxConnections , nurons[I].currnetConections, td < stats.maxConnections - nurons[I].currnetConections);
            if (td < stats.maxConnections-nurons[I].currnetConections) {
                nurons[I].currnetConections++;
                if (nurons[I].currnetConections > stats.maxConnections) return;
                for (unsigned i = 0; i < nurons[I].currnetConections; i++)
                {
                    if (!nurons[I].connections[i].connected)//if the the connection status is false create a connection
                    {
                        nurons[I].connections[i].connected = true;
                        // ana rbitry starting number
                        nurons[I].connections[i].time = 10;
                        // new connection id  =  z*area + y * maxXDimentid + x 

                        nurons[I].connections[i].id = (zd * (stats.volume.y * stats.volume.x)) + (yd * stats.volume.x) + xd;
                        i = nurons[I].currnetConections;
                        //debug printf("!NEW!->I:%u:||C#:%u||CId:%u||x:%u||y:%u||z:%u||Calk:%u||con#:%u\n", I, i, nurons[I].connections[i].id, xd, yd, zd, (zd * (stats.volume.y * stats.volume.x)) + (yd * stats.volume.x) + xd, nurons[I].currnetConections);

                    }
                    /*debug else
                    {
                        printf("I:%u:||C#:%u||CId:%u\n", I, i, nurons[I].connections[i].id);
                    }*/

                }
                //printf("----------------!NEW!->I:%u:||C#:%u||CId:%u||x:%u||y:%u||z:%u||Calk:%u||con#:%u\n", I, i, nurons[I].connections[i].id, xd, yd, zd, (zd * (stats.volume.y * stats.volume.x)) + (yd * stats.volume.x) + xd, nurons[I].currnetConections);

            }

            /*   if (x<0 || y<0 || z<0 || x>stats.volume.x || y>stats.volume.y || z>stats.volume.z) {
                //printf("connection started, but failed. connection will not be set.\n");
                //x = nurons[I].position.x;
                //y = nurons[I].position.y;
                //z = nurons[I].position.Z;
            }
            else if (true)
            {

                //printf("test");
                //incremnt number of connections
                nurons[I].currnetConections++;

                for (unsigned i = 0; i < nurons[I].currnetConections; i++)
                {
                    if (!nurons[I].connections[i].connected)//if the the connection status is false create a connection
                    {
                        nurons[I].connections[i].connected = true;
                        // ana rbitry starting number
                        nurons[I].connections[i].time = 10;
                        // new connection id  =  z*area + y * maxXDimentid + x 
                        nurons[I].connections[i].id = (z * (stats.volume.y * stats.volume.x)) + (y * stats.volume.x) + x;
                        i = nurons[I].currnetConections;
                    }

                }

            }*/
            
        }
        
    }
    
     hipError_t Nurons::save(std::string fileName) {
        std::cout << "saving to file" << std::endl;
        unsigned int theadsPerBlock = 1024;

        std::string outpuit;
        // nurons
        outpuit =
            (std::string)"Nurons\n" +
            "total: " + std::to_string(stats.totalNum) + '\n' +
            "shape: " + std::to_string(stats.shape) + '\n' +
            "volume: " + std::to_string(stats.volume.x) + "," + std::to_string(stats.volume.y) + "," + std::to_string(stats.volume.z) + '\n' +
            "maxConnections: " + std::to_string(stats.maxConnections)+ '\n' +
            "maxRange: " + std::to_string(stats.maxRange);

        // allconnections gpu acceloration;

       /*
        Nurons
        total: #
        shape: #
        volume: X,Y,Z
        maxConnections: X
        maxSensitiity:
        maxNuronTime:
        maxRange:

        Nuron: # -----------------------8  + maxIdDigits
        position: #, #, # --------------11 + (maxIdDigits*3)+2
        Range: # -----------------------8  + maxRangeDigits
        sensitivity: # -----------------14 + maxSensitivityDigits
        curCons: -----------------------10 + maxconnectionsdigs
        activation: # ------------------13 + 1
        sensor: b ----------------------9  + 1
        currnetConnectionBool: ********-24 + (connectiosPerNuron-1)+(connectiosPerNuron)
        currentConnectionTime: ********-24 + (connectiosPerNuron-1)+(connectiosPerNuron*maxYimeDigits)
        currnetConnectionID: ********---22 + (connectiosPerNuron-1)+(connectiosPerNuron*maxIdDigits)


        Nuron : # + 1
        *
        *
        *
        *
        *
        */

        //ptr to all nurond on gpu+
        Nuron* dev_allNur = {};
        /* //ptr to nurons calss on gpu
        Nuron* dev_Nurs;*/
        //ptr to all conectiond on the gpu
        connection* dev_allconects = {};
        //ptr to the Nurons calss
        Nurons* dev_nuronsClass;
        //ptr to device momory holding the nuron decription c~ctring
        char* dev_NuronDecription = {};
        //ptr to device stats
        STE::sts* dev_stats;


        //ptr to nuron decription on host
        char* host_NuronDecription = {};

        unsigned int maxIdDigits = (unsigned int)log10(stats.totalNum) + 1;
        unsigned int maxSensitivityDigits = ((unsigned int)log10(stats.maxSensitivity) + 1);
        unsigned int MaxTimeDigits = ((unsigned int)log10(stats.maxNuronTime) + 1);
        unsigned int connectiosPerNuron = ((unsigned int)log10(stats.maxConnections) + 1);
        unsigned int maxRangeDigits = ((unsigned int)log10(stats.maxRange) + 1);

        unsigned int infoPerNuron =
            9 + maxIdDigits +//id
            11 + (maxIdDigits * 3) + 2 +//pos
            8 + maxRangeDigits +//range
            14 + maxSensitivityDigits +//sensitivity
            10 + connectiosPerNuron +
            13 + 1 +//activation
            11 + 1 +//isSensor
            17 + (stats.maxConnections - 1) + (stats.maxConnections) + // connections is connected ()bool
            17 + (stats.maxConnections - 1) + (stats.maxConnections * MaxTimeDigits) + //time of each connection
            15 + (stats.maxConnections - 1) + (stats.maxConnections * maxIdDigits);//id of each connection

        /*std::cout << " 8 + maxIdDigits = " << 8 + maxIdDigits << std::endl;
        std::cout << " 11 + (maxIdDigits * 3) + 2 = " << 11 + (maxIdDigits * 3) + 2 << std::endl;
        std::cout << "  8 + maxRangeDigits = " << 8 + maxRangeDigits << std::endl;
        std::cout << " 14 + maxSensitivityDigits = " << 14 + maxSensitivityDigits << std::endl;
        std::cout << " 13 + 1 = " << 13 + 1 << std::endl;
        std::cout << " 11 + 1 = " << 11 + 1 << std::endl;
        std::cout << " 17 + (connectiosPerNuron - 1) + (connectiosPerNuron) = " << 17 + (connectiosPerNuron - 1) + (connectiosPerNuron) << std::endl;
        std::cout << " 17 + (connectiosPerNuron - 1) + (connectiosPerNuron * MaxTimeDigits) = " << 17 + (connectiosPerNuron - 1) + (connectiosPerNuron * MaxTimeDigits) << std::endl;
        std::cout << " 15 + (connectiosPerNuron - 1) + (connectiosPerNuron * maxIdDigits) = " << 17 + (connectiosPerNuron - 1) + (connectiosPerNuron * maxIdDigits) << std::endl;
        */

        unsigned int host_NuronDecriptionSize = ((stats.totalNum * infoPerNuron) + 1);
        host_NuronDecription = (char*)malloc(host_NuronDecriptionSize * sizeof(char));



        //unsigned int threds = 64;
        dim3 grid((stats.totalNum / theadsPerBlock) + 1, 1, 1);

        
        std::ofstream myfile;

        //cuda error vareable
        hipError_t cudaStatus = hipSetDevice(0);
        if (errors.isErr()) 
        {
            std::cout << "bob";
            goto ErrorSAVE;
        }


        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto ErrorSAVE;
        }
        /*
        //allocate space on the gpu for all the nurons and set the dev_allNur ptr to that location
        cudaStatus = hipMalloc((void**)&dev_stats, stats.totalNum * sizeof(STE::sts));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrorSAVE;
        }

        //Copy nurons to device memory
        cudaStatus = hipMemcpy(dev_stats, &stats, stats.totalNum * sizeof(STE::sts), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrorSAVE;
        }*/

        //allocate space on the gpu for all the nurons and set the dev_allNur ptr to that location
        cudaStatus = hipMalloc((void**)&dev_allNur, stats.totalNum * sizeof(STE::Nuron));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrorSAVE;
        }

        //Copy nurons to device memory
        cudaStatus = hipMemcpy(dev_allNur, allNurons, stats.totalNum * sizeof(STE::Nuron), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrorSAVE;
        }

        //allocate meemory on device for all the connections
        cudaStatus = hipMalloc((void**)&dev_allconects, stats.totalNum * stats.maxConnections * sizeof(connection));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrorSAVE;
        }

        //coppy all cenections to device memory
        cudaStatus = hipMemcpy(dev_allconects, allConections, stats.totalNum * stats.maxConnections * sizeof(connection), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrorSAVE;
        }


        /* //allocate space on gpu for the Nurons Object NOT the Nurons them selves(the Nuron class)
          cudaStatus = hipMalloc((void**)&dev_Nurs, sizeof(Nurons));
          if (cudaStatus != hipSuccess) {
              fprintf(stderr, "hipMalloc failed!");
              goto ErrorSAVE;
          }

          //coppy from the Nurons cals sto the device
          cudaStatus = hipMemcpy(dev_Nurs, this, sizeof(Nurons), hipMemcpyHostToDevice);
          if (cudaStatus != hipSuccess) {
              fprintf(stderr, "hipMemcpy failed!");
              goto ErrorSAVE;
          }*/


          //allocate space on the gpu for the text to be stored in
        cudaStatus = hipMalloc((void**)&dev_NuronDecription, host_NuronDecriptionSize * sizeof(char));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrorSAVE;
        }




        // Launch a kernel on the GPU with one thread for each element.

        SaveKernal << < grid, theadsPerBlock >> > (dev_NuronDecription, infoPerNuron, stats, dev_allNur, dev_allconects);



        // Check for any errors launching the kernelwwww
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto ErrorSAVE;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto ErrorSAVE;
        }

        cudaStatus = hipMemcpy(host_NuronDecription, dev_NuronDecription, host_NuronDecriptionSize * sizeof(char), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!\n");
            goto ErrorSAVE;
        }
        //std::cout << "end save" << std::endl;
        goto NoErrorSave;
    
    NoErrorSave:
        host_NuronDecription[host_NuronDecriptionSize - 1] = NULL;
        //std::cout << "-----------------------------------------------------------------------------------" << std::endl;
        if (errors.isErr())
        
        std::cout << "saving";
        outpuit = outpuit + host_NuronDecription;// +std::string(host_NuronDecription);
      
        myfile.open(fileName);
        myfile << outpuit;
        //std::cout << outpuit << std::endl;
        myfile.close();
                
        //std::cout << "\n\n\n\n\n\n\n\ncpu result \n" << host_NuronDecription << std::endl;
        hipFree(dev_allNur);
        hipFree(dev_allconects);
        hipFree(dev_NuronDecription);
        free(host_NuronDecription);
        printf("Save function end\n");
        
        if (cudaStatus != hipSuccess) {
            if (!errors.isErr())errors.push(cudaStatus);
        }
        return cudaStatus;
    ErrorSAVE:
        std::cout << "error saving: " << std::endl;

        myfile.close();
        errors.reedAll();
        std::cout << hipGetErrorString(cudaStatus) << std::endl;

        hipFree(dev_allNur);
        hipFree(dev_allconects);
        hipFree(dev_NuronDecription);
        free(host_NuronDecription);
        printf("Save function end\n");
        return cudaStatus;

    }

    __global__ void SaveKernal(char* nuronDecription, unsigned int infoPerNuron, STE::sts stats, STE::Nuron* nurons, connection* conections) {
        //unsigned int I = (blockIdx.x * blockDim.x) + threadIdx.x;
        //printf("nuron: %d\n", I);
        unsigned int I = (blockIdx.x * blockDim.x) + threadIdx.x;
        //loop through  maxDigitsId number of time (EX:4)
        if (I >= stats.totalNum) {
            //printf("extra id %i", I);

        }
        else {
            //printf("%u\n", I);
            //printf("extra id %i \n", I);


            //unsigned int position = I * infoPerNuron;
            /*
                Nuron: # -----------------------8 + maxIdDigits
                position: #, #, # --------------11 + (maxIdDigits*3)+2
                Range: # -----------------------8 + maxRangeDigits
                sensitivity: # -----------------14 + maxSensitivityDigits
                curCons: -----------------------11 + maxconnectionsdigs
                activation: # ------------------13 + 1
                sensor: b ----------------------9+1
                currnetConnectionBool: ********-24 + ((connectiosPerNuron*2)-1)
                currentConnectionTime: ********-24 + (connectiosPerNuron-1)+(connectiosPerNuron*maxYimeDigits)
                currnetConnectionID: ********---22 + (connectiosPerNuron-1)+(connectiosPerNuron*maxIdDigits)


                Nuron : # + 1
                *
                *
                *
                *
            */
            // decide the number of digits to use when writing position and id
            unsigned int maxDigitsId = (unsigned int)(log10((double)stats.totalNum) + 1);
            //number of digits 
            unsigned int digitsID;
            //a simple int to char converter
            char digits[] = { '0', '1', '2', '3', '4', '5', '6', '7', '8', '9' };
            //current ptr to where we want to write text
            char* ptrStart;
            //vvvvvv trouble shooting
            ptrStart = nuronDecription + (I * infoPerNuron);

            //^^^^^^^^^^
            //temporary vareable to hold text
            char* tempVar = "\n\nNuron: ";

            //print id
            // a temporary vareable to be used in the loop if the max number of digits in the id is 4 (EX: 8593) we set maxdigitsIdTemp to 1000
            unsigned int maxdigitsIdTemp;

            //vvvvvvvvvvuncomment in the kenal
           // ptrStart = nuronDecription + (I * infoPerNuron);
            //^^^^^^^^^^


            digitsID = (unsigned int)log10((double)I) + 1;

            //coppy tempvar to the current pointed at adress 8 chariters long (8 chariters because {\n,N,u,r,o,n,:, }is 8 chariters)  -- tecnicaly there is a null pter at the end of tempvar but we dont want that coppied
            memcpy(ptrStart, tempVar, 9);
            //move the ptr addres 8 char units to pass the writtentext  
            ptrStart = ptrStart + 9;


            //set to max id digits
            maxdigitsIdTemp = (unsigned int)pow((double)10, (double)(maxDigitsId - 1));



            for (unsigned i = 0; i < maxDigitsId; i++)
            {
                /* at this point we wright each digit as a chariter.
                * ath the begining of the fires loop we have the following (with new line charters being shown before the newline itselv being displayeed happening)
                * "\n
                * Nuron: "
                * we also have maxdigitsIdTemp = 10^maxDigitsId (EX:10000)
                * and I is arbitrary (EX: 1500)
                */

                //after this line the next unused chariter in the memory loation pointed to by PtrStart is set to I/maxdigitsIdTemp (in our example this works out to 1500/1000 = 1 ...integers round down...)
                //we take the modulo to clamp numbers from 0 to 9 so we dont acess out side of the digits char array.
                //digits is used to change a single digit base 10 number to a single char 

                ptrStart[i] = digits[(I / maxdigitsIdTemp) % 10];
                //std::cout << "I: " << I << "\nmaxdigitsIdTemp: " << maxdigitsIdTemp << std::endl;  <--for testing
                //std::cout << ptrStart[i] << std::endl;                                             <--for testing
                //after this line we remove a zeo from maxdigitsIdTemp. using this metod we cycle through the whole id chariter by cahriter
                maxdigitsIdTemp = maxdigitsIdTemp / 10;

                // this is repeeted
                // in this method we get leading zeros witch i am fine with this means with an I of 50 and a max digits of 4  we get 0050
            }



            //we now move the ptrstart location maxdigitsID number of units witch leaves  ptrStart[]
            ptrStart = ptrStart + maxDigitsId;

            //print position x,y,z
            //the same method is repeted for x,y,and z values as the id number
            tempVar = "\nposition: ";
            memcpy(ptrStart, tempVar, 11);
            ptrStart = ptrStart + 11;

            //SAVE Xpos
            //std::cout << "Xpos: ";

            maxdigitsIdTemp = (unsigned int)pow((double)10, (double)(maxDigitsId - 1));
            for (unsigned i = 0; i < maxDigitsId; i++)//print id
            {
                ptrStart[i] = digits[(nurons[I].position.x / maxdigitsIdTemp) % 10];
                maxdigitsIdTemp = maxdigitsIdTemp / 10;
                //std::cout << ptrStart[i] << std::endl;                                            

            }
            ptrStart[maxDigitsId] = ',';
            ptrStart = ptrStart + maxDigitsId + 1;



            //SAVE ypos
            maxdigitsIdTemp = (unsigned int)pow((double)10, (double)(maxDigitsId - 1));
            for (unsigned i = 0; i < maxDigitsId; i++)//print id
            {
                ptrStart[i] = digits[(nurons[I].position.y / maxdigitsIdTemp) % 10];
                maxdigitsIdTemp = maxdigitsIdTemp / 10;
            }
            ptrStart[maxDigitsId] = ',';
            ptrStart = ptrStart + maxDigitsId + 1;

            //SAVE Zpos
            maxdigitsIdTemp = (unsigned int)pow((double)10, (double)(maxDigitsId - 1));
            for (unsigned i = 0; i < maxDigitsId; i++)//print id
            {
                ptrStart[i] = digits[(nurons[I].position.z / maxdigitsIdTemp) % 10];
                maxdigitsIdTemp = maxdigitsIdTemp / 10;
            }
            ptrStart = ptrStart + maxDigitsId;

            //std::cout << nuronDecription << std::endl;

            //print range
            //agan we use the same method but maxDigitsIdid is recalulated with the maximum range use to calulate the number of digits
            tempVar = "\nRange: ";
            memcpy(ptrStart, tempVar, 8);
            ptrStart = ptrStart + 8;
            maxDigitsId = (unsigned int)(log10((double)stats.maxRange) + 1);
            maxdigitsIdTemp = (unsigned int)pow((double)10, (double)(maxDigitsId - 1));

            for (unsigned i = 0; i < maxDigitsId; i++)
            {
                ptrStart[i] = digits[(stats.maxRange / maxdigitsIdTemp) % 10];
                maxdigitsIdTemp = maxdigitsIdTemp / 10;
            }
            ptrStart = ptrStart + maxDigitsId;



            //print sensitivity
            //agan we use the same method but maxDigitsIdid is recalulated with the maximum sensitivity use to calulate the number of digits
            tempVar = "\nsensitivity: ";
            memcpy(ptrStart, tempVar, 14);
            ptrStart = ptrStart + 14;

            maxDigitsId = (unsigned int)(log10((double)stats.maxSensitivity) + 1);
            maxdigitsIdTemp = (unsigned int)pow((double)10, (double)(maxDigitsId - 1));
            for (unsigned i = 0; i < maxDigitsId; i++)
            {
                ptrStart[i] = digits[(stats.maxSensitivity / maxdigitsIdTemp) % 10];
                maxdigitsIdTemp = maxdigitsIdTemp / 10;
            }
            ptrStart = ptrStart + maxDigitsId;

            //print current connections
           //agan we use the same method but maxDigitsIdid is recalulated with the maximum sensitivity use to calulate the number of digits
            tempVar = "\ncurCons: ";
            memcpy(ptrStart, tempVar, 10);
            ptrStart = ptrStart + 10;

            maxDigitsId = (unsigned int)(log10((double)stats.maxConnections) + 1);
            maxdigitsIdTemp = (unsigned int)pow((double)10, (double)(maxDigitsId - 1));
            for (unsigned i = 0; i < maxDigitsId; i++)
            {
                //printf("curcons: %d", digits[(stats.maxConnections / maxdigitsIdTemp) % 10]);
                ptrStart[i] = digits[(stats.maxConnections / maxdigitsIdTemp) % 10];
                maxdigitsIdTemp = maxdigitsIdTemp / 10;
            }
            ptrStart = ptrStart + maxDigitsId;/**/

            //print its activation (1= active,  0 = not active)
            //here it is simple cast the bool activation to an unsigned int the store the value 
            tempVar = "\nactivation: ";
            memcpy(ptrStart, tempVar, 13);
            ptrStart = ptrStart + 13;
            ptrStart[0] = digits[unsigned(nurons[I].activation) % 10];
            ptrStart = ptrStart + 1;


            //print if sensor (1= yes, 0 = no)
            tempVar = "\nisSensor: ";
            memcpy(ptrStart, tempVar, 11);
            ptrStart = ptrStart + 11;
            ptrStart[0] = digits[unsigned(nurons[I].sensor) % 10];
            ptrStart = ptrStart + 1;

            //print if nuron is connection conected (1 = conected = true, 0 = not)
            tempVar = "\ncurrentConBool: ";
            memcpy(ptrStart, tempVar, 17);
            ptrStart = ptrStart + 17;

            for (unsigned i = 0; i < stats.maxConnections; i++)
            {
                ptrStart[i * 2] = digits[(unsigned)(conections[stats.maxConnections * I + i].connected) % 10];
                //printf("\n %u,%u: %u", I,i, conections[stats.maxConnections * I + i].connected);
                ptrStart[(i * 2) + 1] = ',';
            }

            ptrStart = ptrStart + (2 * stats.maxConnections) - 1;


            //print connected nuron time
            tempVar = "\ncurrentConTime: ";
            memcpy(ptrStart, tempVar, 17);
            ptrStart = ptrStart + 17;


            maxDigitsId = (unsigned int)(log10((double)stats.maxNuronTime) + 1);


            for (unsigned i = 0; i < stats.maxConnections; i++)
            {
                maxdigitsIdTemp = pow((double)10, (double)(maxDigitsId - 1));
                for (unsigned j = 0; j < maxDigitsId; j++)
                {
                    //ptrStart[(i * (maxDigitsId + 1)) + j] = digits[(nurons->connections->time / maxdigitsIdTemp)%9];
                    ptrStart[j] = digits[(conections[stats.maxConnections * I + i].time / maxdigitsIdTemp) % 10];
                    maxdigitsIdTemp = maxdigitsIdTemp / 10;
                }
                ptrStart[maxDigitsId] = ',';
                ptrStart = ptrStart + maxDigitsId + 1;
            }

            ptrStart = ptrStart - 1;

            //print connected nuron ID
            tempVar = "\ncurrentConId: ";
            memcpy(ptrStart, tempVar, 15);
            ptrStart = ptrStart + 15;

            maxDigitsId = (unsigned int)(log10((double)stats.totalNum) + 1);

            for (unsigned i = 0; i < stats.maxConnections; i++)
            {
                maxdigitsIdTemp = (unsigned int)pow((double)10, (double)(maxDigitsId - 1));
                //printf("save:: Nur%u: (%u = %u)\n", I, i, conections[stats.maxConnections * I + i].id);
                for (unsigned j = 0; j < maxDigitsId; j++)
                {
                    ptrStart[j] = digits[(conections[stats.maxConnections * I + i].id / maxdigitsIdTemp) % 10];
                    
                    maxdigitsIdTemp = maxdigitsIdTemp / 10;
                }

                //printf("\n");
                if (i + 1 < stats.maxConnections)
                {
                    ptrStart[maxDigitsId] = ',';
                    ptrStart = ptrStart + 1;

                }
                
                ptrStart = ptrStart + maxDigitsId;
            }
        }
    }

     hipError_t Nurons::load(std::string path) {

        unsigned int theadsPerBlock = 1024;

        //ptr to all nurond on gpu+
        Nuron* dev_allNur = {};
        connection* dev_allconects = {};
        char* dev_filText = {};

       
       
     
        //loads file headder
          //file opening magic
        //http://www.cplusplus.com/forum/beginner/229845/
        std::ifstream file{ path };
        std::string const str = static_cast<std::ostringstream> (std::ostringstream{} << file.rdbuf()).str();

        hipError_t cudaStatus = hipSetDevice(0);
       
       
            //set token to look for 
        std::string find = "total:";//<<make a way of modifing thees throu a save rule set
            //look for token
        unsigned int offset = STE::parse::parseForLoc(find.data(), find.size(), str.data(), str.size()) + find.size() + 1;
            //save vlaue to memory
        unsigned int total = STE::parse::getUnsigned(str.data() + offset, str.size() - offset);

            //repeat
        find = "shape:";
        offset = STE::parse::parseForLoc(find.data(), find.size(), str.data(), str.size()) + find.size() + 1;
        unsigned int shape = STE::parse::getUnsigned(str.data() + offset, str.size() - offset);

        find = "volume:";
        offset = STE::parse::parseForLoc(find.data(), find.size(), str.data(), str.size()) + find.size() + 1;
        unsigned int volX = STE::parse::getUnsigned(str.data() + offset, str.size() - offset);

        find = ",";
        offset = STE::parse::parseForLoc(find.data(), find.size(), str.data(), str.size()) + find.size() + 1;
        unsigned int volY = STE::parse::getUnsigned(str.data() + offset, str.size() - offset);

        find = ",";
        offset = offset + STE::parse::parseForLoc(find.data(), find.size(), str.data() + offset, str.size() + offset) + find.size() + 1;
        unsigned int volZ = STE::parse::getUnsigned(str.data() + offset, str.size() - offset);
        dim3 vol(volX, volY, volZ);

        find = "maxConnections:";
        offset = STE::parse::parseForLoc(find.data(), find.size(), str.data(), str.size()) + find.size() + 1;
        unsigned int maxConnections = STE::parse::getUnsigned(str.data() + offset, str.size() - offset);
        if (total = !vol.x * vol.y * vol.z) {
            return cudaStatus;
        }

        find = "maxRange:";
        offset = STE::parse::parseForLoc(find.data(), find.size(), str.data(), str.size()) + find.size() + 1;
        unsigned int maxRange = STE::parse::getUnsigned(str.data() + offset, str.size() - offset);
        if (total = !vol.x * vol.y * vol.z) {
            return cudaStatus;
        }

        //find the start of the first nuron description.||later crate an error handler for parse so that errors can be detetced.
        
        find = "maxRange:";
        offset = STE::parse::parseForLoc(find.data(), find.size(), str.data(), str.size()) + find.size();
        find = "Nuron:";
        offset = offset + STE::parse::parseForLoc(find.data(), find.size(), str.data() + offset, str.size() - offset) -1;
      
        const char* NurDecStart = str.data() + offset;

        //find the end of the first nuron decription
        find = "currentConId:";
        offset = STE::parse::parseForLoc(find.data(), find.size(), str.data(), str.size()) + find.size() + 1;
        find = "\n";
        offset = offset + STE::parse::parseForLoc(find.data(), find.size(), str.data() + offset, str.size() + offset) + find.size();
        const char* endNurDes = str.data() + offset;
       
        //apply headder to curreent nurrons
        //temporary vareables
        Nuron* tempNurPtr;
        connection* tempConnPtr;
        //create a "Nurons" object with the paramiters from file
        STE::Nurons temp(maxConnections, maxRange, vol, shape);//<<using 1 for nuron range it will be changed later oin code.
        //coppy stats object from temp
        this->stats = temp.stats;

        if (errors.isErr()) goto ErrorLoad;

        if (str.data() == 0) {
            std::cout << "no save file found" << std::endl;
            //add err stuff add cutom errors
            errors.push(cudaStatus);
            goto ErrorLoad;
        }

        //point to memory from the original Nurons object 
        tempNurPtr = this->allNurons;
        tempConnPtr = this->allConections;

        //have original nurons object adopt the temp memory
        this->allNurons = temp.allNurons;
        this->allConections = temp.allConections;

        //have the temp nurons object adopt he data from the temp pointers 
        temp.allConections = tempConnPtr;
        temp.allNurons = tempNurPtr;
        //htis^^^ section crates a temporary Nurons object with the recorded paramaters in order to initalize the data.
        //and then this section swapps the data from the original object and the new object. 
        //this alows for the old data to be deallocated cleenly within the new object.


        //unsigned int threds = 64;
        dim3 grid((stats.totalNum / theadsPerBlock) + 1, 1, 1);


        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto ErrorLoad;
        }

        //allocate space on the gpu for all the nurons and set the dev_allNur ptr to that location
        cudaStatus = hipMalloc((void**)&dev_allNur, stats.totalNum * sizeof(STE::Nuron));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrorLoad;
        }
        //Copy nurons to device memory
        cudaStatus = hipMemcpy(dev_allNur, allNurons, stats.totalNum * sizeof(STE::Nuron), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrorLoad;
        }

        //allocate meemory on device for all the connections
        cudaStatus = hipMalloc((void**)&dev_allconects, stats.totalNum * stats.maxConnections * sizeof(connection));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrorLoad;
        }
        //coppy all cenections to device memory
        cudaStatus = hipMemcpy(dev_allconects, allConections, stats.totalNum * stats.maxConnections * sizeof(connection), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrorLoad;
        }

          //allocate space on the gpu for the text to be stored in
        cudaStatus = hipMalloc((void**)&dev_filText, (str.size()-(str.data()- NurDecStart)) * sizeof(char));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrorLoad;
        }

        cudaStatus = hipMemcpy(dev_filText, NurDecStart, (str.size() - (str.data() - NurDecStart)) * sizeof(char), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrorLoad;
        }


        // Launch a kernel on the GPU with one thread for each element.

        LoadKernal <<< grid, theadsPerBlock >>> (dev_filText, (endNurDes- NurDecStart),stats, dev_allNur, dev_allconects);



        // Check for any errors launching the kernelwwww
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto ErrorLoad;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto ErrorLoad;
        }

        // Copy nurons from gpu memory to cpu memory.
        cudaStatus = hipMemcpy(allNurons, dev_allNur, stats.totalNum * sizeof(STE::Nuron), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrorLoad;
        }

        // Copy connections from gpu memory to cpu memory.
        cudaStatus = hipMemcpy(allConections, dev_allconects, stats.totalNum* stats.maxConnections * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrorLoad;
        }

        //on gpu sat values for nurons;
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
            goto ErrorLoad;
        }

    ErrorLoad:
    
        hipFree(dev_allNur);
        hipFree(dev_allconects);
        hipFree(dev_filText);
       
        if (cudaStatus != hipSuccess) {
            if (!errors.isErr())errors.push(cudaStatus);
        }
        return cudaStatus;
    }
     
    __global__ void LoadKernal(char* dev_filText, unsigned int nurBlock, STE::sts stats, STE::Nuron* nurons, connection* connections) {
        /* 
        * Initilize
        * *textPtr1
        * 
        * set text ptr to the location of the relevent nuron
        * 
        */
        //nuron id
        
        unsigned int I = (blockIdx.x * blockDim.x) + threadIdx.x;
        //chech id dosent exceed number of nur
        if (I >= stats.totalNum) {
            //printf("extra id %i", I);
        }
        else {
            
            //current ptr to where we want to write text
            //move the startiung ptr to the start of the corisponding block
           // prob wont need this -> char* currentPtr 
            const char* startPtr = dev_filText + (I * (nurBlock));
            unsigned int offset = 0;
            //grab ans set stats
            //test check
            
            //printf("nurblock %d \n", I);
            /*if (I = 0)
            {
                printf("chariter: %c, nuron: \n", startPtr[2], I);
            }*/
            
            
           
               // printf("%d, %d, %c \n", nurons[I].id, STE::parseDev::getUnsigned(startPtr, nurBlock), (startPtr+2)[0]);
            

            //range -- make a function to do this (gpu runable)
            char* find = "Range:";
            unsigned findLen = 6;

           
            STE::parseDev::parseForLoc(find, findLen, startPtr, nurBlock);
            offset = STE::parseDev::parseForLoc(find, findLen, startPtr, nurBlock) + findLen;
            nurons[I].range = STE::parseDev::getInt(startPtr + offset, nurBlock - offset);
            
            //sensitivity
            find = "sensitivity:";
            findLen = 12;
            offset = STE::parseDev::parseForLoc(find, findLen, startPtr, nurBlock) + findLen;
            nurons[I].sensitivity = STE::parseDev::getInt(startPtr + offset, nurBlock - offset);

            //activation 
            find = "activation:";
            findLen = 11;
            offset = STE::parseDev::parseForLoc(find, findLen, startPtr, nurBlock) + findLen;
            nurons[I].activation = STE::parseDev::getInt(startPtr + offset, nurBlock - offset);
            
            //sensor
            find = "isSensor:";
            findLen = 9;
            offset = STE::parseDev::parseForLoc(find, findLen, startPtr, nurBlock) + findLen;
            nurons[I].sensor = (bool)STE::parseDev::getInt(startPtr + offset, nurBlock - offset);

            //current con bool (multy)
            find = "currentConBool:";
            findLen = 15;
            offset = STE::parseDev::parseForLoc(find, findLen, startPtr, nurBlock) + findLen;
            if (stats.maxConnections > 0) {// check math on this
                connections[stats.maxConnections * I ].connected = (bool)STE::parseDev::getInt(startPtr + offset, nurBlock - offset);
                if (stats.maxConnections > 1) {
                    for (unsigned i = 1; i < stats.maxConnections;i++)
                    {

                        find = ",";
                        offset = STE::parseDev::parseForLoc(find, findLen, startPtr, nurBlock) + findLen;
                        connections[stats.maxConnections * I + i].connected = (bool)STE::parseDev::getInt(startPtr + offset, nurBlock - offset);
                    }
                }
            }

           //current con time (multy)
            find = "currentConTime:";
            findLen = 15;
            offset = STE::parseDev::parseForLoc(find, findLen, startPtr, nurBlock) + findLen;
            if (stats.maxConnections > 0) {// check math on this
                connections[stats.maxConnections * I].connected = (bool)STE::parseDev::getInt(startPtr + offset, nurBlock - offset);
                if (stats.maxConnections > 1) {
                    for (unsigned i = 1; i < stats.maxConnections; i++)
                    {
                        find = ",";
                        offset = STE::parseDev::parseForLoc(find, findLen, startPtr, nurBlock) + findLen;
                        connections[stats.maxConnections * I + i].time = STE::parseDev::getInt(startPtr + offset, nurBlock - offset);
                    }
                }
            }

            //current con ID (multy)
            find = "currentConId:";
            findLen = 13;
            offset = STE::parseDev::parseForLoc(find, findLen, startPtr, nurBlock) + findLen;
            if (stats.maxConnections > 0) {// check math on this
                connections[stats.maxConnections * I].connected = (bool)STE::parseDev::getInt(startPtr + offset, nurBlock - offset);
                if (stats.maxConnections > 1) {
                    for (unsigned i = 1; i < stats.maxConnections; i++)
                    {
                        find = ",";
                        offset = STE::parseDev::parseForLoc(find, findLen, startPtr, nurBlock) + findLen;
                        connections[stats.maxConnections * I + i].id = STE::parseDev::getInt(startPtr + offset, nurBlock - offset);
                    }
                }
                
            }
            
            //nurons[I].sensitivity = STE::parseDev::getInt(startPtr + offset, nurBlock - offset);
            
        }
    }

     hipError_t Nurons::setNuronSens(unsigned quant, unsigned* ids, bool sensor) {

        unsigned int theadsPerBlock = 1024;

        dim3 grid((stats.totalNum / theadsPerBlock) + 1, 1, 1);

        Nuron* dev_allNur = {};
        unsigned* dev_ids = {};

        //cuda error vareable
        hipError_t cudaStatus = hipSetDevice(0);
        if (errors.isErr()) goto ErrSetSens;
       

        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto ErrSetSens;
        }
       
        //allocate space on the gpu for all the nurons and set the dev_allNur ptr to that location
        cudaStatus = hipMalloc((void**)&dev_allNur, stats.totalNum * sizeof(STE::Nuron));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrSetSens;
        }

        //Copy nurons to device memory
        cudaStatus = hipMemcpy(dev_allNur, allNurons, stats.totalNum * sizeof(STE::Nuron), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrSetSens;
        }

        cudaStatus = hipMalloc((void**)&dev_ids, quant * sizeof(unsigned));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrSetSens;
        }

        //Copy nurons to device memory
        cudaStatus = hipMemcpy(dev_ids, ids, quant *sizeof(unsigned), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrSetSens;
        }

        // Launch a kernel on the GPU with one thread for each element.

        setSensKernal <<< grid, theadsPerBlock >>> (dev_allNur, quant,dev_ids, sensor);


        // Check for any errors launching the kernelwwww
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto ErrSetSens;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto ErrSetSens;
        }

        cudaStatus = hipMemcpy(allNurons, dev_allNur, stats.totalNum * sizeof(STE::Nuron), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrSetSens;
        }

        //std::cout << "end save" << std::endl;
        goto noErrSetSens;
    ErrSetSens:

    noErrSetSens:
        //std::cout << "\n\n\n\n\n\n\n\ncpu result \n" << host_NuronDecription << std::endl;
        hipFree(dev_allNur);
        hipFree(dev_ids);
        if (cudaStatus != hipSuccess) {
            if (!errors.isErr())errors.push(cudaStatus);
        }
        return cudaStatus;
     }

    __global__ void setSensKernal(Nuron* nurons, unsigned quant, unsigned* ids, bool sensor) {
        unsigned int I = (blockIdx.x * blockDim.x) + threadIdx.x;
        //loop through  maxDigitsId number of time (EX:4)
        if (I >= quant) {
            return;
        }
        nurons[ids[I]].sensor = sensor;
    }

    hipError_t Nurons::updateSens(unsigned quant, unsigned* ids, bool* data) {
        unsigned int theadsPerBlock = 1024;
        dim3 grid((stats.totalNum / theadsPerBlock) + 1, 1, 1);

        Nuron* dev_allNur = {};
        unsigned* dev_ids = {};
        bool* dev_data = {};

        //cuda error vareable
        hipError_t cudaStatus = hipSetDevice(0);
        if (errors.isErr()) goto ErrSetSenUpdate;

        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto ErrSetSenUpdate;
        }

        //allocate space on the gpu for all the nurons and set the dev_allNur ptr to that location
        cudaStatus = hipMalloc((void**)&dev_allNur, stats.totalNum * sizeof(STE::Nuron));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrSetSenUpdate;
        }

        //Copy nurons to device memory
        cudaStatus = hipMemcpy(dev_allNur, allNurons, stats.totalNum * sizeof(STE::Nuron), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrSetSenUpdate;
        }

        cudaStatus = hipMalloc((void**)&dev_ids, quant * sizeof(unsigned));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrSetSenUpdate;
        }

        //Copy nurons to device memory
        cudaStatus = hipMemcpy(dev_ids, ids, quant * sizeof(unsigned), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrSetSenUpdate;
        }


        cudaStatus = hipMalloc((void**)&dev_data, quant * sizeof(bool));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrSetSenUpdate;
        }

        //Copy nurons to device memory
        cudaStatus = hipMemcpy(dev_data, ids, quant * sizeof(bool), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrSetSenUpdate;
        }
        // Launch a kernel on the GPU with one thread for each element.

        updateSensKernal <<< grid, theadsPerBlock >> > (dev_allNur, quant, dev_ids, dev_data);


        // Check for any errors launching the kernelwwww
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto ErrSetSenUpdate;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto ErrSetSenUpdate;
        }

        cudaStatus = hipMemcpy(allNurons, dev_allNur, stats.totalNum * sizeof(STE::Nuron), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrSetSenUpdate;
        }

        //std::cout << "end save" << std::endl;
        goto noErrSetSenUpdate;
    ErrSetSenUpdate:

    noErrSetSenUpdate:
        //std::cout << "\n\n\n\n\n\n\n\ncpu result \n" << host_NuronDecription << std::endl;
        hipFree(dev_allNur);
        hipFree(dev_data);
        hipFree(dev_ids);

        if (cudaStatus != hipSuccess) {
            if (!errors.isErr())errors.push(cudaStatus);
        }

        return cudaStatus;
    }

    __global__ void updateSensKernal(Nuron* nurons, unsigned quant, unsigned* ids, bool* data) {
        unsigned int I = (blockIdx.x * blockDim.x) + threadIdx.x;
        //loop through  maxDigitsId number of time (EX:4)
        if (I >= quant) {
            return;
        }
        nurons[ids[I]].activation = data[I];
    }
     
    hipError_t Nurons::updateOut(unsigned quant, unsigned* ids, bool* data) {
        unsigned int theadsPerBlock = 1024;
        dim3 grid((stats.totalNum / theadsPerBlock) + 1, 1, 1);

        Nuron* dev_allNur = {};
        unsigned* dev_ids = {};
        bool* dev_data = {};

        //cuda error vareable
        hipError_t cudaStatus = hipSetDevice(0);
        if (errors.isErr()) goto ErrOutDataUpdate;
        

        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto ErrOutDataUpdate;
        }

        //allocate space on the gpu for all the nurons and set the dev_allNur ptr to that location
        cudaStatus = hipMalloc((void**)&dev_allNur, stats.totalNum * sizeof(STE::Nuron));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrOutDataUpdate;
        }

        //Copy nurons to device memory
        cudaStatus = hipMemcpy(dev_allNur, allNurons, stats.totalNum * sizeof(STE::Nuron), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrOutDataUpdate;
        }

        cudaStatus = hipMalloc((void**)&dev_ids, quant * sizeof(unsigned));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrOutDataUpdate;
        }

        //Copy nurons to device memory
        cudaStatus = hipMemcpy(dev_ids, ids, quant * sizeof(unsigned), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrOutDataUpdate;
        }


        cudaStatus = hipMalloc((void**)&dev_data, quant * sizeof(bool));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto ErrOutDataUpdate;
        }

        //Copy nurons to device memory
        cudaStatus = hipMemcpy(dev_data, ids, quant * sizeof(bool), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrOutDataUpdate;
        }
        // Launch a kernel on the GPU with one thread for each element.

        updateOutKernal <<< grid, theadsPerBlock >> > (dev_allNur, quant, dev_ids, dev_data);

        // Check for any errors launching the kernelwwww
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto ErrOutDataUpdate;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto ErrOutDataUpdate;
        }

        cudaStatus = hipMemcpy(data, dev_data, quant * sizeof(bool), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto ErrOutDataUpdate;
        }

        //std::cout << "end save" << std::endl;
        goto noErrOutDataUpdate;
    ErrOutDataUpdate:

    noErrOutDataUpdate:
        //std::cout << "\n\n\n\n\n\n\n\ncpu result \n" << host_NuronDecription << std::endl;
        hipFree(dev_allNur);
        hipFree(dev_data);
        hipFree(dev_ids);
        
        if (cudaStatus != hipSuccess) {
            if (!errors.isErr())errors.push(cudaStatus);
        }
        return cudaStatus;
    }

    __global__ void updateOutKernal(Nuron* nurons, unsigned quant, unsigned* ids, bool* data) {
        unsigned int I = (blockIdx.x * blockDim.x) + threadIdx.x;
        //loop through  maxDigitsId number of time (EX:4)
        if (I >= quant) {
            return;
        }
        data[I] = nurons[ids[I]].activation;
    }

    hipError_t Rand(int** data, unsigned quantity, int range, int offset) //reruns a pointer to allo=cated memmory containg random numbers within the range and ofset
    {
        //comments remove latter
        // 
        // 
        //printf("range:%d", range);
        //ptr to all nurond on gpu
        int* results;
        //cuda error vareable
        hipError_t cudaStatus = hipSetDevice(0);
        
        //dim3 RNG3(rand() % 100 + 900000, rand() % 100 + 900000, rand() % 100 + 900000);

        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto Test1Error;
        }

        // allocate space on the gpu forresults
        cudaStatus = hipMalloc((void**)&results, quantity * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Test1Error;
        }
        //seed neds to change fix

        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
//        srand(time(NULL));
        //uint64_t ms = std::chrono::duration_cast<std::chrono::milliseconds>
        uint64_t ms = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
//        hiprandSetPseudoRandomGeneratorSeed(gen, 7487 * rand());
        //debug std::cout << ms << std::endl;
        hiprandSetPseudoRandomGeneratorSeed(gen, 7487 * ms);
        hiprandGenerate(gen, ((unsigned*)results), quantity);
       
        // Launch a kernel on the GPU with one thread for each element.
         // create a grid of threds with the quantity of all the nurons
        unsigned int theadsPerBlock = 1024;
        dim3 grid((quantity / theadsPerBlock) + 1, 1, 1);
        rad <<< grid, theadsPerBlock >> > (results,quantity, range, offset);

        // Check for any errors launching the kernelwwww
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Test1Error;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto Test1Error;
        }

        // Copy nurons from gpu memory to cpu memory.
        //free(data);//this seems wrong
       // *data = (int*)malloc(quantity * sizeof(int));

        cudaStatus = hipMemcpy(*data, results, quantity * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Test1Error;
        }
        
      // printf("host rng:%d,%d,%d,%d-------\n", (*data)[0], (*data)[1], (*data)[2], (*data)[3]);

       /*debug for (size_t i = 0; i < quantity; i++)
        {
            std::cout << "value[" << i << "]:" << (*data)[i] << std::endl;
        }*/

    Test1Error:
    
        hipFree(results);
        return cudaStatus;
    
    }

    //optomize rand by adding a boolian to the function to handle the destination ptr as a device or host ptr.

    __global__ void rad(int* data, unsigned quantity ,unsigned range, int offset) {
        
        unsigned int I = (blockIdx.x * blockDim.x) + threadIdx.x;
        //chech id dosent exceed number of nur
        if (I >= quantity) {
            return;
            //printf("extra id %i", I);
        }
        //try hiprand
        //debug printf("I:%u - Data:%i = %i mod %u + %i\n", I,data[I] % range + offset, data[I], range, offset);
        data[I] =( data[I] % range + offset);
        
        //printf("Data: %i\n", data[I]);
    }
}

extern "C"
{


}









//https://docs.nvidia.com/cuda/hiprand/host-api-overview.html
#endif







//Repressed================================================================================================
/*
    __global__ void updateKernal(STE::Nuron* nurons, connection* conections, sts stats, int* rng) {// only works when all nurons have the same number of max connections-

       //initilization---------------------------------------------------
        unsigned int I = (blockIdx.x * blockDim.x) + threadIdx.x;
        //end start--------------------------------------------------

        int timeAdd = 2;
        int timeSub = 1;
        int sensAdd = 2;
        int sensSub = 1;

        int* rngX;
        int* rngY;
        int* rngZ;
        int* rngT;

        rngX = rng + (3 * I);
        rngY = rngX + 1;
        rngZ = rngY + 1;
        rngT = rng + (3 * stats.totalNum) + I;

        //printf("rngX:%d\nrngY:%d\nrngZ:%d\nrngT:%d\n\n", rngX, rngY, rngZ, rngT);
        if (stats.totalNum < I) return;//exit if excee
        //if (I != 0)return;
        nurons[I].connections = conections + (I * stats.maxConnections);
        //if (I == 0)printf("1Device current connect:%d\n", nurons[1].currnetConections);


        //summ all conceted nuron conections----------------------------------------------------------------------------------
        int fialValue = 0;
        for (unsigned i = 0; i < stats.maxConnections; i++) // accelorate this
        {
            if (nurons[I].connections[i].connected)
            {
                if (nurons[nurons[I].connections[i].id].activation)
                {
                    fialValue = fialValue + (nurons[I].connections[i].time);
                    if (nurons[I].connections[i].time += timeAdd > nurons[I].maxNuronTime)
                    {
                        nurons[I].connections[i].time = nurons[I].maxNuronTime;
                    }

                }
                else
                {
                    //decreemt time  left and check if time is zeroif so remove from list
                    if (nurons[I].connections[i].time -= timeSub < 1) {
                        nurons[I].connections[i].connected = false;
                        nurons[I].connections[i].time = 0;
                        nurons[I].currnetConections--;
                    }
                }
            }
        }

        //if (I == 0)printf("2Device current connect:%d\n", nurons[1].currnetConections);

        // activate or not-------------------------------------------------------------------------------------------
        if (!nurons[I].sensor)
        {
            if (fialValue > nurons[I].sensitivity)
            {
                nurons[I].activation = true;
                nurons[I].sensitivity += sensAdd;
            }
            else {
                nurons[I].activation = false;
                nurons[I].sensitivity += sensSub;
            }

            if (nurons[I].sensitivity > nurons[I].maxSensitivity)
            {
                nurons[I].sensitivity = nurons[I].maxSensitivity;
            }

        }
        else {
            //write to sensor
            //nurons[I].activation = true;
        }


        //cpmpare a random number to the amount of un used connections----------------------------------------------------------------------------------------------------------
        rngX = rng + (3 * I);
        rngY = rngX + 1;
        rngZ = rngY + 1;
        rngT = rng + (3 * stats.totalNum) + I;

        if(*rngT> (nurons[I].currnetConections / stats.maxConnections))
        {
            if (nurons[I].currnetConections>0) {
                printf("got 2");
            }


            unsigned int xLow = nurons[I].range;
            unsigned int xHigh = xLow;
            unsigned int yLow = xLow;
            unsigned int yHigh = xLow;
            unsigned int zLow = xLow;
            unsigned int zHigh = xLow;

            //make shure selection is in the volume of nurons
            if (nurons[I].position.x + xHigh > stats.volume.x)
            {
                xHigh = stats.volume.x - nurons[I].position.x;
            }

            if ((int)(nurons[I].position.x - xLow) < 0)
            {
                xLow = nurons[I].position.x;
            }

            if (nurons[I].position.y + yHigh > stats.volume.y)
            {
                yHigh = stats.volume.y - nurons[I].position.y;
            }

            if ((int)(nurons[I].position.y - yLow) < 0)
            {
                yLow = nurons[I].position.y;
            }

            if (nurons[I].position.z + zHigh > stats.volume.z)
            {
                zHigh = stats.volume.z - nurons[I].position.z;
            }

            if ((int)(nurons[I].position.z - zLow) < 0)
            {
                zLow = nurons[I].position.z;
            }

            //choos a location
            int x = nurons[I].position.x + *rngX % (xLow + 1 + xHigh) - xLow;
            int y = nurons[I].position.y + *rngY % (xLow + 1 + yHigh) - yLow;
            int z = nurons[I].position.z + *rngZ % (zLow + 1 + zHigh) - zLow;


            for (unsigned i = 0; i < nurons[I].currnetConections; i++)
            {
                if (!nurons[I].connections[i].connected)
                {

                    nurons[I].connections[i].connected = true;
                    // ana rbitry starting number
                    nurons[I].connections[i].time = 500;
                    // new connection id  =  z*area + y * maxXDimentid + x
                    nurons[I].connections[i].id = (z * (stats.volume.y * stats.volume.x)) + (y * stats.volume.x) + x;
                   // printf("is:%d  \n", I);
                    //incremnt number of connections
                    nurons[I].currnetConections++;
                    i = stats.maxConnections;
                    printf("connecting(%d)to(%d-{%d,%d,%d})\n", I, (z* (stats.volume.y* stats.volume.x)) + (y * stats.volume.x) + x,x,y,z);
                }
            }
        }
        //addconnections
        //goal probability of forming a new conection decreses with number of currnet connections.
    }
    */