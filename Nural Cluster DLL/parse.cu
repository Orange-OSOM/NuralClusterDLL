#ifndef ParseC
#define ParseC

#include <iostream>
#include "hip/hip_runtime.h"
#include ""

#include "parse.h"

namespace STE
{
	unsigned int parse::parseForQuant(const char* token, unsigned int tokelen, const char* sourse, unsigned int soursLen) {
		unsigned int tokenCharInd = 0;
		unsigned int quantity = 0;
		for (unsigned int i = 0; i < soursLen; i++)
		{
			if (sourse[i] == token[tokenCharInd])
			{
				tokenCharInd++;
				if (tokenCharInd == tokelen) {
					//end of corect word
					tokenCharInd = 0;
					quantity++;
				}
			}
			else {
				tokenCharInd = 0;
			}
		}
		return quantity;
	}

	unsigned int parse::parseForLoc(const char* token, unsigned int tokelen, const char* sourse, unsigned int soursLen) /*returns index of first chariter of token*/ {
		unsigned int tokenCharInd = 0;
		unsigned int posOfTok = 0;

		for (unsigned int i = 0; i < soursLen; i++)
		{

			if (sourse[i] == token[tokenCharInd]) {
				tokenCharInd++;
				if (tokenCharInd == tokelen) {
					//end of corect word
					posOfTok = i;
					i = soursLen;
				}
			}
			else {
				tokenCharInd = 0;
			}
		}
		//if (posOfTok - tokelen < 0) return soursLen;
		return posOfTok - tokelen;
	}

	char* parse::parseForLocPTR(const char* token, unsigned int tokelen, const char* sourse, unsigned int soursLen) {
		return (char*)(sourse + parseForLoc(token, tokelen, sourse, soursLen));
	}

	unsigned int parse::getUnsigned(const char* sourse, unsigned int soursLen) {
		unsigned int whiteSpaces = 0;
		unsigned int result = 0;
		//move past all non numarical chariters
		while (sourse[whiteSpaces] < '0' || sourse[whiteSpaces] > '9' && whiteSpaces < soursLen) {

			whiteSpaces++;
		}
		// translate the chariter limit to use less math in loop
		soursLen = soursLen - whiteSpaces;
		sourse = sourse + whiteSpaces;
		//loop to chariter limit or to end of number

		for (unsigned int i = 0; i < soursLen; i++)
		{
			if (sourse[i] >= '0' && sourse[i] <= '9')
			{
				result = (result * 10) + (sourse[i] - '0');
			}
			else
			{
				break;
			}
		}
		return result;
	}

	int getInt(char* sourse, unsigned int soursLen) {
		unsigned int whiteSpaces = 0;
		unsigned int result = 0;
		bool positive = true;
		//move past all non numarical chariters
		while (((sourse[whiteSpaces] < '0' && sourse[whiteSpaces] > '9') || (sourse[whiteSpaces] == '-')) && whiteSpaces < soursLen) {
			whiteSpaces++;
		}
		// translate the chariter limit to use less math in loop
		soursLen = soursLen - whiteSpaces;
		sourse = sourse + whiteSpaces;

		//loop to chariter limit or to end of number
		if (sourse[0] == '-') {
			whiteSpaces = 0;
			positive = false;
			while (sourse[whiteSpaces] < '0' && sourse[whiteSpaces] > '9' && whiteSpaces < soursLen) {
				{
					whiteSpaces++;
				}
				soursLen = soursLen - whiteSpaces;
				sourse = sourse + whiteSpaces;
			}

			for (unsigned int i = 0; i < soursLen; i++)
			{
				if (sourse[i] >= '0' && sourse[i] <= '9')
				{
					result = (result * 10) + (sourse[i] - '0');
				}
				else
				{
					break;
				}
			}
			if (!positive) {
				result = -1 * result;
			}
			return result;
		}

		/*float getFloat(char* sourse, unsigned int soursLen){}*/
	}

	
	
	/*
	__device__ unsigned int parseDev::parseForQuant(const char* token, unsigned int tokelen, const char* sourse, unsigned int soursLen) {
		unsigned int tokenCharInd = 0;
		unsigned int quantity = 0;
		for (unsigned int i = 0; i < soursLen; i++)
		{
			if (sourse[i] == token[tokenCharInd])
			{
				tokenCharInd++;
				if (tokenCharInd == tokelen) {
					//end of corect word
					tokenCharInd = 0;
					quantity++;
				}
			}
			else {
				tokenCharInd = 0;
			}
		}
		return quantity;
	}

	__device__ unsigned int parseDev::parseForLoc(const char* token, unsigned int tokelen, const char* sourse, unsigned int soursLen)  { //returns index of first chariter of token
		unsigned int tokenCharInd = 0;
		unsigned int posOfTok = 0;

		for (unsigned int i = 0; i < soursLen; i++)
		{

			if (sourse[i] == token[tokenCharInd]) {
				tokenCharInd++;
				if (tokenCharInd == tokelen) {
					//end of corect word
					posOfTok = i;
					i = soursLen;
				}
			}
			else {
				tokenCharInd = 0;
			}
		}
		return posOfTok - tokelen;
	}

	__device__ char* parseDev::parseForLocPTR(const char* token, unsigned int tokelen, const char* sourse, unsigned int soursLen) {
		return (char*)(sourse + parseForLoc(token, tokelen, sourse, soursLen));
	}

	__device__ unsigned int parseDev::getUnsigned(const char* sourse, unsigned int soursLen) {
		unsigned int whiteSpaces = 0;
		unsigned int result = 0;
		//move past all non numarical chariters
		while (sourse[whiteSpaces] < '0' || sourse[whiteSpaces] > '9' && whiteSpaces < soursLen) {

			whiteSpaces++;
		}
		// translate the chariter limit to use less math in loop
		soursLen = soursLen - whiteSpaces;
		sourse = sourse + whiteSpaces;
		//loop to chariter limit or to end of number

		for (unsigned int i = 0; i < soursLen; i++)
		{
			if (sourse[i] >= '0' && sourse[i] <= '9')
			{
				result = (result * 10) + (sourse[i] - '0');
			}
			else
			{
				break;
			}
		}
		return result;
	}
		
	__device__ int parseDev::getInt(const char* sourse, unsigned int soursLen) {
		unsigned int whiteSpaces = 0;
		unsigned int result = 0;
		bool positive = true;
		//move past all non numarical chariters
		while (((sourse[whiteSpaces] < '0' && sourse[whiteSpaces] > '9') || (sourse[whiteSpaces] == '-')) && whiteSpaces < soursLen) {
			whiteSpaces++;
		}
		// translate the chariter limit to use less math in loop
		soursLen = soursLen - whiteSpaces;
		sourse = sourse + whiteSpaces;

		//loop to chariter limit or to end of number
		if (sourse[0] == '-') {
			whiteSpaces = 0;
			positive = false;
			while (sourse[whiteSpaces] < '0' && sourse[whiteSpaces] > '9' && whiteSpaces < soursLen) {
				{
					whiteSpaces++;
				}
				soursLen = soursLen - whiteSpaces;
				sourse = sourse + whiteSpaces;
			}

			for (unsigned int i = 0; i < soursLen; i++)
			{
				if (sourse[i] >= '0' && sourse[i] <= '9')
				{
					result = (result * 10) + (sourse[i] - '0');
				}
				else
				{
					break;
				}
			}
			if (!positive) {
				result = -1 * result;
			}
			return result;
		}

		//float getFloat(char* sourse, unsigned int soursLen){}
	}

	__device__  int parseDev::testint() { return 0; }*/
}


namespace pog {

	__device__  int testin::testint() { return 0; }
}

#endif